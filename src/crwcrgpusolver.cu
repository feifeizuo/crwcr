#include "hip/hip_runtime.h"
#include <algorithm>
#include"crwcrkernel.cuh"
#include "crwcrgpusolver.h"

#pragma comment(lib, "cudart.lib")

CRWCRSolver::CRWCRSolver(const float * image, int width, int height) :
	image_(image),
	width_(width),
	height_(height),
	seeds_(nullptr),
	parameters_(Singleton<Parameters>::GetInstance())
{
	numPixels_ = width_ * height_;
	
	hipMalloc((void**)&d_wx, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_wy, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_grad, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_seedBuffer_, numPixels_ * sizeof(unsigned char));
	hipMalloc((void**)&d_matSub, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_matCen, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_matUp, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_rVec, numPixels_ * sizeof(float));
	hipMalloc((void**)&d_solution, numPixels_ * sizeof(float));

	solution_ = new float[numPixels_];

	calculateWeight();
	calculateGradient();
}

CRWCRSolver::~CRWCRSolver()
{
	hipFree(d_grad);
	hipFree(d_wx);
	hipFree(d_wy);
	hipFree(d_solution);
	hipFree(d_matCen);
	hipFree(d_matUp);
	hipFree(d_matSub);
	hipFree(d_rVec);
	hipFree(d_seedBuffer_);
	delete[] solution_;
	solution_ = nullptr;
}

void CRWCRSolver::setSeed(TwoLabelSeed * seed)
{
	seeds_ = seed;
}

void CRWCRSolver::solve()
{
	hipMemcpy(d_seedBuffer_, seeds_->getSeedBuffer(), numPixels_ * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	const int blockDim = 8;

	dim3 dimBlock(blockDim, blockDim, 1);
	dim3 dimGrid(CRWCRGPU::iDivUp(width_,blockDim), CRWCRGPU::iDivUp(height_, blockDim), 1);

	dim3 dimB(blockDim, 1, 1);
	dim3 dimG(CRWCRGPU::iDivUp(width_*height_, blockDim), 1, 1);

	int2 dim = make_int2(width_, height_);

	for(int iter=0;iter<parameters_.maxIterations1D;iter++)
	{
		CRWCRGPU::rowSweepKernel << <dimGrid, dimBlock >> > (d_wx, d_grad,
			d_seedBuffer_, d_matSub, d_matCen, d_matUp, d_rVec,dim,parameters_.gamma1D,parameters_.lambda1D);
		CRWCRGPU::TDMARow << <dimG, dimB >> > (d_matSub, d_matCen, d_matUp, d_rVec, d_solution, dim);
		CRWCRGPU::increaseSeedsKernel<<<dimGrid, dimBlock>>>(d_solution, d_seedBuffer_,dim, parameters_.foreThreshold);
		
		CRWCRGPU::columnSweepKernel<< <dimGrid, dimBlock >> > (d_wy, d_grad,
			d_seedBuffer_, d_matSub, d_matCen, d_matUp, d_rVec, dim, parameters_.gamma1D, parameters_.lambda1D);
		CRWCRGPU::TDMAColumn << <dimG, dimB >> > (d_matSub, d_matCen, d_matUp, d_rVec, d_solution, dim);
		CRWCRGPU::increaseSeedsKernel<<<dimGrid, dimBlock>>>(d_solution, d_seedBuffer_,dim, parameters_.foreThreshold);
	}

	for (int iter = 0; iter < parameters_.maxIterations2D; iter++)
	{
		CRWCRGPU::PRRowKernel << <dimGrid, dimBlock >> > (d_wx, d_wy, d_grad,
			d_solution, d_seedBuffer_, d_matSub, d_matCen, d_matUp, d_rVec,dim,parameters_.gamma2D,
			parameters_.lambda2D,parameters_.dt);
		CRWCRGPU::TDMARow << <dimG, dimB >> > (d_matSub, d_matCen, d_matUp, d_rVec, d_solution, dim);

		CRWCRGPU::PRColumnKernel << <dimGrid, dimBlock >> > (d_wx, d_wy, d_grad,
			d_solution, d_seedBuffer_, d_matSub, d_matCen, d_matUp, d_rVec
			, dim, parameters_.gamma2D,	parameters_.lambda2D, parameters_.dt);
		CRWCRGPU::TDMAColumn << <dimG, dimB >> > (d_matSub, d_matCen, d_matUp, d_rVec, d_solution, dim);
	}

	hipMemcpy(solution_, d_solution, numPixels_ * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	time_ = time;
	
}

float * CRWCRSolver::generateProbabilityImage() const
{
	return solution_;
}

int CRWCRSolver::getUseTime()const
{
	return time_;
}

void CRWCRSolver::normalize(float * data, size_t length)
{
	float l = 1, u = 0;
	for (size_t i = 0; i < length; i++)
	{
		l = std::min(l, data[i]);
		u = std::max(u, data[i]);
	}

	if (fabs(l - u) < 1e-6)
	{
		return;
	}

	for (size_t i = 0; i < length; i++)
		data[i] = (data[i] - l) / (u - l);
}

void CRWCRSolver::calculateWeight()
{
	const float beta = 80, epsilon = 1e-5;
	float* weight = new float[numPixels_];
	memset(weight,0,numPixels_*sizeof(float));
	
	for (int y = 0; y < height_; y++)
	{
		for (int x = 0; x < width_ - 1; x++)
		{
			int index = y * width_ + x;
			weight[index] = (fabs((image_[index] - image_[index + 1])));
		}
	}

	normalize(weight, numPixels_);

	for (size_t i = 0; i < numPixels_; i++)
	{
		weight[i] = exp(-beta * weight[i]) + epsilon;
	}

	hipMemcpy(d_wx, weight, numPixels_*sizeof(float),hipMemcpyHostToDevice);

	memset(weight, 0, numPixels_ * sizeof(float));
	for (int x = 0; x < width_; x++)
	{
		for (int y = 0; y < height_ - 1; y++)
		{
			// a better store order
			weight[x * height_ + y] = fabs((image_[y * width_ + x] - image_[(y + 1) * width_ + x]));
		}
	}

	normalize(weight, numPixels_);
	for (size_t i = 0; i < numPixels_; i++)
	{
		weight[i] = exp(-beta * weight[i]) + epsilon;
	}

	hipMemcpy(d_wy, weight, numPixels_*sizeof(float),hipMemcpyHostToDevice);

	delete[] weight;
	weight=nullptr;
}

void CRWCRSolver::calculateGradient()
{
	float *grad = new float[numPixels_];
	memset(grad, 0, numPixels_ * sizeof(float));

	for (size_t x = 1; x < width_ - 1; x++)
	{
		for (size_t y = 1; y < height_ - 1; y++)
		{
			float gx = image_[x - 1 + y * width_] - image_[x + 1 + y * width_];
			float gy = image_[x + (y - 1) * width_] - image_[x + (y + 1) * width_];

			grad[x + y * width_] = fabs(gx) + fabs(gy);

		}
	}
	normalize(grad, numPixels_);

	hipMemcpy(d_grad, grad, numPixels_*sizeof(float),hipMemcpyHostToDevice);

	delete[] grad;
	grad=nullptr;
}
